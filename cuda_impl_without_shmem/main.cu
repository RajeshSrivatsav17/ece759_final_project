#include <iostream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include "parameters.h"
#include "Utilities.h"
#include "buoyantforce.h"
#include "advect.h"
#include "divergence.h"
#include "cga.cuh"
#include "boundary_cond.h"
#include "hip/hip_runtime.h"
#include "velocity_correction.h"

int main(){
    
    hipEvent_t startEvent, stopEvent;
    hipEvent_t startEvent_totalSteps, stopEvent_totalSteps;
    float elapsedTime;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventCreate(&startEvent_totalSteps);
    hipEventCreate(&stopEvent_totalSteps);

    size_t totalSize = XDIM * YDIM * ZDIM;

    using array_t = float (&) [XDIM][YDIM][ZDIM];
    float *uRaw = new float [totalSize]; //Velocity in x direction
    float *vRaw = new float [totalSize]; //Velocity in y direction
    float *wRaw = new float [totalSize]; //Velocity in z direction
    float *rhoRaw = new float [totalSize]; //Density
    float *TRaw = new float [totalSize]; //Temperature
    float *divergenceRaw = new float [totalSize]; //Divergence
    float *pRaw = new float [totalSize]; //Pressure

    float* uRaw_star = new float[totalSize];
    float* vRaw_star = new float[totalSize];
    float* wRaw_star = new float[totalSize];
    float* rhoRaw_next = new float[totalSize];
    float* TRaw_next = new float[totalSize];
    int totalSteps = 500;

    float *uRaw_d, *vRaw_d, *wRaw_d;             // Velocity components
    float *rhoRaw_d, *TRaw_d;                    // Density and Temperature
    float *divergenceRaw_d, *pRaw_d;             // Divergence and Pressure
    
    // Allocate on device
    hipMalloc((void**)&uRaw_d, totalSize*sizeof(float));
    hipMalloc((void**)&vRaw_d, totalSize*sizeof(float));
    hipMalloc((void**)&wRaw_d, totalSize*sizeof(float));
    hipMalloc((void**)&rhoRaw_d, totalSize*sizeof(float));
    hipMalloc((void**)&TRaw_d, totalSize*sizeof(float));
    hipMalloc((void**)&divergenceRaw_d, totalSize*sizeof(float));
    hipMalloc((void**)&pRaw_d, totalSize*sizeof(float));

    float *uRaw_star_d, *vRaw_star_d, *wRaw_star_d;
    float *rhoRaw_next_d, *TRaw_next_d;
    
    hipMalloc((void**)&uRaw_star_d, totalSize * sizeof(float));
    hipMalloc((void**)&vRaw_star_d, totalSize * sizeof(float));
    hipMalloc((void**)&wRaw_star_d, totalSize * sizeof(float));
    hipMalloc((void**)&rhoRaw_next_d, totalSize * sizeof(float));
    hipMalloc((void**)&TRaw_next_d, totalSize * sizeof(float));
    
    //Velocity//
    array_t u = reinterpret_cast<array_t>(*uRaw); //Velocity in x direction
    array_t v = reinterpret_cast<array_t>(*vRaw); //Velocity in y direction
    array_t w = reinterpret_cast<array_t>(*wRaw); //Velocity in z direction
    //Density//
    array_t rho = reinterpret_cast<array_t>(*rhoRaw);
    //Temperature//
    array_t T = reinterpret_cast<array_t>(*TRaw);
    //Divergence// 
    array_t divergence = reinterpret_cast<array_t>(*divergenceRaw);
    //Pressure//
    array_t p = reinterpret_cast<array_t>(*pRaw);

    //Advection Velocity//
    array_t u_star = reinterpret_cast<array_t>(*uRaw_star); //Velocity in x direction
    array_t v_star = reinterpret_cast<array_t>(*vRaw_star); //Velocity in y direction
    array_t w_star = reinterpret_cast<array_t>(*wRaw_star); //Velocity in z direction
    //Advection Density//
    array_t rho_star = reinterpret_cast<array_t>(*rhoRaw_next);
    //Advection Temperature//
    array_t T_star = reinterpret_cast<array_t>(*TRaw_next);

    Clear(u);Clear(w);Clear(divergence);Clear(p);
    Clear(u_star);Clear(v_star);Clear(w_star);
    
    InitializeProblem(rho,T,v);

    hipMemcpy(uRaw_d, uRaw, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(vRaw_d, vRaw, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(wRaw_d, wRaw, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(rhoRaw_d, rhoRaw, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(TRaw_d, TRaw, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(divergenceRaw_d, divergenceRaw, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(pRaw_d, pRaw, totalSize, hipMemcpyHostToDevice);

    hipMemcpy(uRaw_star_d, uRaw_star, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(vRaw_star_d, vRaw_star, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(wRaw_star_d, wRaw_star, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(rhoRaw_next_d, rhoRaw_next, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(TRaw_next_d, TRaw_next, totalSize, hipMemcpyHostToDevice);

    int threadsPerBlock = 512; // 8*8*8 tile
    int blocksPerGrid = (totalSize + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(startEvent_totalSteps, 0);

    for (int t = 0; t < totalSteps; ++t) {
        hipEventRecord(startEvent, 0);
        // Step 1
        //std::cout<<"Calling buoyantforce()\n";
        buoyantforce_kernel<<<blocksPerGrid, threadsPerBlock>>>(rhoRaw_d,TRaw_d,vRaw_d); //applying buoyant force on pressure and temperature of smoke from vertical velocity compoenent
        hipDeviceSynchronize();
        //std::cout<<"Returned from buoyantforce()\n";
        // Step 2: Advect velocity (u*, v*, w*)
        //std::cout<<"Calling semi_lag_adv() for u\n";
        semi_lagrangian_advection_kernel<<<blocksPerGrid, threadsPerBlock>>>(uRaw_star_d, uRaw_d, uRaw_d, vRaw_d, wRaw_d, dt);
        //std::cout<<"Returned from semi_lag_adv() for u\n";
        //std::cout<<"Calling semi_lag_adv() for v\n";
        semi_lagrangian_advection_kernel<<<blocksPerGrid, threadsPerBlock>>>(vRaw_star_d, vRaw_d, uRaw_d, vRaw_d, wRaw_d, dt);
        //std::cout<<"Returned from semi_lag_adv() for v\n";
        //std::cout<<"Calling semi_lag_adv() for w\n";
        semi_lagrangian_advection_kernel<<<blocksPerGrid, threadsPerBlock>>>(wRaw_star_d, wRaw_d, uRaw_d, vRaw_d, wRaw_d, dt);
        //std::cout<<"Returned from semi_lag_adv() for w\n";
        // Step 2: Advect smoke density and temperature
        //std::cout<<"Calling semi_lag_adv() for rho\n";
        semi_lagrangian_advection_kernel<<<blocksPerGrid, threadsPerBlock>>>(rhoRaw_next_d, rhoRaw_d, uRaw_d, vRaw_d, wRaw_d, dt);
        //std::cout<<"Returned from semi_lag_adv() for rho\n";
        //std::cout<<"Calling semi_lag_adv() for Temp\n";
        semi_lagrangian_advection_kernel<<<blocksPerGrid, threadsPerBlock>>>(TRaw_next_d, TRaw_d, uRaw_d, vRaw_d, wRaw_d, dt);
        //std::cout<<"Returned from semi_lag_adv() for Temp\n";

        // Swap buffers for next timestep
        //std::cout<<"Calling swap buffer()\n";
        std::swap(uRaw_d, uRaw_star_d);
        std::swap(vRaw_d, vRaw_star_d);
        std::swap(wRaw_d, wRaw_star_d);
        std::swap(rhoRaw_d, rhoRaw_next_d);
        std::swap(TRaw_d, TRaw_next_d);

        //std::cout<<"Finished swapping\n";
        // Step 3: Divergence of velocity
        //std::cout<<"Calling Divergence()\n";
        computeDivergence_kernel<<<blocksPerGrid,threadsPerBlock>>>(uRaw_d, vRaw_d, wRaw_d, divergenceRaw_d);
        hipDeviceSynchronize();
        //std::cout<<"Returned from Divergence()\n";
        // Step 4: Iterative solver
        //std::cout<<"Calling CG()\n";
        solvePressureCG(pRaw_d, divergenceRaw_d);
        //std::cout<<"Returned from CG()\n";
        // Step 5: Velocity correction
        //std::cout<<"Velocity correction initiated()\n";
        velocityCorrection_kernel<<<blocksPerGrid,threadsPerBlock>>>(uRaw_d, vRaw_d, wRaw_d, pRaw_d);
        //std::cout<<"Velocity Correction done\n";
        // Step 6: Boundary Condition
        //std::cout<<"Calling boundary()\n";
        applyBoundaryConditions(uRaw_d,vRaw_d,wRaw_d);
        //std::cout<<"Returned from boundary()\n";
        //if (t % 10 == 0)
        //    writetoCSV(rho, "density_frame_" + std::to_string(t) + ".csv","density");
        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
        std::cout << "CUDA Event compute time for one frame #  " << t << " = " 
                  << elapsedTime / 1000.0f << " sec\n";        
    }
    hipEventRecord(stopEvent_totalSteps,0);
    hipEventSynchronize(stopEvent_totalSteps);

    hipEventElapsedTime(&elapsedTime, startEvent_totalSteps, stopEvent_totalSteps);
    std::cout << "\n\nCUDA Event compute time for " << totalSteps << " frames  " << elapsedTime / 1000.0f << " sec\n";  
    delete[] uRaw;
    delete[] vRaw;
    delete[] wRaw;
    delete[] rhoRaw;
    delete[] TRaw;
    delete[] divergenceRaw;
    delete[] pRaw;

    return 0;
}